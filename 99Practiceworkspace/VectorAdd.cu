/* Author Deeksha Prakash Kankalale
   Date : August 3 2025
   Description : CPP host code with vector 1D. 

   LeetGPU : Question 1
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>

using namespace std;

__global__ void VectorAdd(float *a, float *b, float* c, size_t sizeV)
{
    int tid = threadIdx.x +blockIdx.x * blockDim.x;
    c[tid]= a[tid] + b[tid];
}

int main()
{
    vector<float> A(100),B(100),C(100); //defined vectors in cpp
    size_t sizeVector = 100;    // choose the size of vectors.

    //fill host vector A and B
    for(int i=0; i<sizeVector; i++){
        A[i]=i;
        B[i]=i*2;
    }
    
    //Allocate device vectors
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeVector * sizeof(float));
    hipMalloc((void**)&d_B, sizeVector * sizeof(float));
    hipMalloc((void**)&d_C, sizeVector * sizeof(float));

    //populate the device memory with host memory
    hipMemcpy(d_A, A.data(), sizeVector*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), sizeVector*sizeof(float), hipMemcpyHostToDevice);

    VectorAdd<<<1,100>>>(d_A, d_B, d_C, sizeVector);
    hipDeviceSynchronize();
    //copy the result vector from device to host
    hipMemcpy(C.data(), d_C, sizeVector*sizeof(float), hipMemcpyDeviceToHost);

    //print the result

    for(int j=0; j<sizeVector; j++){
        cout<<A[j]<<" + "<<B[j]<<" = "<<C[j]<<endl;
    }
     
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}
