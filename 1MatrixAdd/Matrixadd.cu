#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void MatrixAdd(const float* A, const float* B, float* C,int N){
    int tid = threadIdx.x + blockIdx.x *blockDim.x;
    if(tid<N){
        C[tid] = A[tid] + B[tid];
    }
}

int main (){  
    // add constant size array
    const int N = 10;
    size_t size = N * sizeof(float);

    //allocate host memory
    float h_A[N], h_B[N], h_C[N];

    for (int i=0; i<N; i++){
        h_A[i]=i;
        h_B[i]=2*i;
    }

    //allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    //copy inputs to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //kernel launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1)/(threadsPerBlock);
    MatrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    //Copy results back to host
     hipMemcpy(h_C,d_C, size, hipMemcpyDeviceToHost);

    //Print the output from the host

    for (int i=0; i<N ; i++){
        printf(" A[%d] %f + B[%d] %f = c[%d] %f \n" ,i,h_A[i], i, h_B[i],i, h_C[i]);
    }
    // cuda free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;

}